
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>
#include <sys/time.h>

__global__ void tick_1s()
{
    for (int i = 0; i < 1e3; ++i)
    {
        __nanosleep(1e6);
    }
}

int main(int argc, char* argv[])
{
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);

    int num_blocks = atoi(argv[1]) * prop.multiProcessorCount;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    tick_1s<<<num_blocks, 1>>>();
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float duration;
    hipEventElapsedTime(&duration, start, stop);

    printf("SM count: %d, use 1 SM to laucnch %d blocks, so ", prop.multiProcessorCount, num_blocks);
    printf("blocks/SM: %d", num_blocks / prop.multiProcessorCount);
    printf(", elapsed time: %.2f\n", duration / 1e3);
}
